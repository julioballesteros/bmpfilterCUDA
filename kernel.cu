#include "hip/hip_runtime.h"
//Realizado por Julio Ballesteros
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "kernel.h"
#define BLOCK_SIZE 512

float mask[] = {
	0.1,0.1,0.1,
	0.1,0.1,0.1,
	0.1,0.1,0.1
};

extern __shared__ uint8 temp[];


__global__ void filtro_k(uint8* img, float* filtro, uint32 imgW, uint32 imgH, uint32 filW, uint32 filH, uint8* imgResult)
{
	
	int gi, gj, lindex, gindex, x, y, threadID;
	uint8 pixOrig;
	uint8 pixResult;

	threadID = blockIdx.x*blockDim.x + threadIdx.x;
	if (threadID >= (imgW*imgH)) return;
	
	gi = threadID / imgW;
	gj = threadID % imgW;

	gindex = gi * imgW + gj;
	lindex = threadIdx.x + imgW + 3;

	temp[lindex] = img[gindex];

	if (threadIdx.x < imgW + 3) {
		if(gindex >= imgW + 3)	temp[lindex - (imgW + 3)] = img[gindex - (imgW + 3)];
		if(gindex < imgW * imgH - (imgW + 3))	temp[lindex + blockDim.x] = img[gindex + blockDim.x];
	}

	if (gi == 0 || gi == imgH - 1)	return;
	if (gj < 3 || gj >= imgW - 3)	return;

	__syncthreads();

	pixOrig = 0;
	pixResult = 0;
	
	for (x = -1; x < 2; x++)
		for (y = -1; y < 2; y++) {
			pixOrig = temp[lindex + x*imgW + y*3];
			pixResult += pixOrig*filtro[(x + 1) + ((y + 1) * 3)];
		}
		
	imgResult[gindex] = pixResult;
}


uint8* applyFilterGPU(uint8* img, float* filtro, uint32 imgW, uint32 imgH, uint32 filW, uint32 filH)
{
	uint8 i, j;
	uint8* img_d = NULL;
	float* filtro_d = NULL;
	uint8* imgResult_d = NULL;
	uint8* imgResult_h = NULL;
	uint8* amplifiedImg = NULL;
	uint8* imgResult = NULL;

	hipMalloc((void**)&img_d, sizeof(uint8)* 3 * (imgW + 2)*(imgH + 2));
	hipMalloc((void**)&imgResult_d, sizeof(uint8)*(imgW + 2)*(imgH + 2) * 3);
	hipMalloc((void**)&filtro_d, sizeof(float)*filH * filW);
	imgResult_h = (uint8*)malloc(sizeof(uint8)*(imgW + 2)*(imgH + 2) * 3);
	amplifiedImg = (uint8*)malloc(3 * (imgW + 2)*(imgH + 2));
	imgResult = (uint8*)malloc(sizeof(uint8)*imgW*imgH * 3);

	memset(amplifiedImg, 0, 3 * (imgW + 2)*(imgH + 2));
	for (i = 1; i <= imgH; i++)
		for (j = 1; j <= imgW; j++) {
			((pixel_t*)amplifiedImg)[j + i * (imgW + 2)] = ((pixel_t*)img)[(j - 1) + (i - 1)*imgW];
		}

	hipMemcpy(img_d, amplifiedImg, sizeof(uint8) * 3 * (imgW + 2)*(imgH + 2), hipMemcpyHostToDevice);
	hipMemcpy(filtro_d, filtro, sizeof(float)*filW * filH, hipMemcpyHostToDevice);

	hipMemset(imgResult_d, 0xFF, sizeof(uint8)*(imgW + 2)*(imgH + 2) * 3);
	memset(imgResult_h, 0xFF, sizeof(uint8)* (imgW + 2)*(imgH + 2) * 3);
	memset(imgResult, 0xFF, sizeof(uint8)* imgW * imgH * 3);

	int numThreadsBloque = BLOCK_SIZE;
	int numBloques = ((imgW + 2)*(imgH + 2)*3) / numThreadsBloque + 1;

	filtro_k << < numBloques, numThreadsBloque, sizeof(uint8) * BLOCK_SIZE + ((imgW + 2 + 1) * 2) * 3 >> > (img_d, filtro_d, (imgW + 2)*3, imgH + 2, filW, filH, imgResult_d);
	hipDeviceSynchronize();

	hipMemcpy(imgResult_h, imgResult_d, sizeof(uint8)*(imgW + 2)*(imgH + 2) * 3, hipMemcpyDeviceToHost);

	for (i = 0; i < imgH; i++)
		for (j = 0; j < imgW; j++) {
			((pixel_t*)imgResult)[j + i * imgW] = ((pixel_t*)imgResult_h)[(j + 1) + (i + 1)*(imgW + 2)];
		}

	hipFree(img_d);
	hipFree(filtro_d);
	hipFree(imgResult_d);
	free(imgResult_h);

	return imgResult;
}

uint8* applyFilterCPU(uint8* img, float* filtro, uint32 imgW, uint32 imgH, uint32 filW, uint32 filH)
{
	uint32  i, j;
	int x, y;
	pixel_t pixOrig;
	pixel_t pixResult;
	uint8* imgResult = NULL;
	uint8* amplifiedImg = NULL;

	imgResult = (uint8*)malloc(sizeof(uint8)*imgW*imgH * 3);
	memset(imgResult, 0xFF, sizeof(uint8)*imgW*imgH * 3);

	amplifiedImg = (uint8*)malloc(3 * (imgW + 2)*(imgH + 2));
	memset(amplifiedImg, 0, 3 * (imgW + 2)*(imgH + 2));
	for (i = 1; i <= imgH; i++)
		for (j = 1; j <= imgW; j++) {
			((pixel_t*)amplifiedImg)[j + i * (imgW + 2)] = ((pixel_t*)img)[(j - 1) + (i - 1)*imgW];
		}

	for (i = 0; i < imgH; i++)
		for (j = 0; j < imgW; j ++) {

			pixOrig.R = pixOrig.G = pixOrig.B = 0;
			pixResult.R = pixResult.G = pixResult.B = 0;

			for (y = -1; y < 2; y++)
				for (x = -1; x < 2; x++)
				{
					pixOrig = ((pixel_t*)amplifiedImg)[(j + x + 1) + ((i + y + 1) * (imgW + 2))];
					pixResult.R += pixOrig.R*filtro[(x + 1) + ((y + 1) * 3)];
					pixResult.G += pixOrig.G*filtro[(x + 1) + ((y + 1) * 3)];
					pixResult.B += pixOrig.B*filtro[(x + 1) + ((y + 1) * 3)];
				}

			((pixel_t*)imgResult)[j + i * imgW] = pixResult;

		}
	return imgResult;
}
